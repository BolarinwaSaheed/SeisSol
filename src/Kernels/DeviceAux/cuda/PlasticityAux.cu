#include "hip/hip_runtime.h"
#include <Kernels/DeviceAux/PlasticityAux.h>
#include <init.h>
#include <cmath>
#include <type_traits>

// NOTE: using c++14 because of cuda@10
namespace seissol {
namespace kernels {
namespace device {
namespace aux {
namespace plasticity {

template<typename T>
__forceinline__ __device__ typename std::enable_if<std::is_floating_point<T>::value, T>::type
squareRoot(T x) {
  return std::is_same<T, double>::value ? sqrt(x) : sqrtf(x);
}

template<typename T>
__forceinline__ __device__ typename std::enable_if<std::is_floating_point<T>::value, T>::type
maxValue(T x, T y) {
  return std::is_same<T, double>::value ? fmax(x, y) : fmaxf(x, y);
}

template<typename Tensor>
__forceinline__  __device__
constexpr size_t leadDim() {
  return Tensor::Stop[0] - Tensor::Start[0];
}

//--------------------------------------------------------------------------------------------------
__global__ void kernel_saveFirstMode(real *firstModes,
                                     const real **modalStressTensors) {
  constexpr auto modalStressTensorsColumn = leadDim<init::Q>();
  firstModes[threadIdx.x + blockDim.x * blockIdx.x] =
      modalStressTensors[blockIdx.x][threadIdx.x * modalStressTensorsColumn];
}

void saveFirstModes(real *firstModes,
                    const real **modalStressTensors,
                    const size_t numElements,
                    void *streamPtr) {
  dim3 block(NUM_STRESS_COMPONENTS, 1, 1);
  dim3 grid(numElements, 1, 1);
  auto stream = reinterpret_cast<hipStream_t>(streamPtr);
  kernel_saveFirstMode<<<grid, block, 0, stream>>>(firstModes, modalStressTensors);
}


//--------------------------------------------------------------------------------------------------
__global__ void kernel_adjustDeviatoricTensors(real **nodalStressTensors,
                                               unsigned *isAdjustableVector,
                                               const PlasticityData *plasticity,
                                               const double oneMinusIntegratingFactor) {
  real *elementTensors = nodalStressTensors[blockIdx.x];
  real localStresses[NUM_STRESS_COMPONENTS];


  constexpr auto elementTensorsColumn = leadDim<init::QStressNodal>();
  #pragma unroll
  for (int i = 0; i < NUM_STRESS_COMPONENTS; ++i) {
    localStresses[i] = elementTensors[threadIdx.x + elementTensorsColumn * i];
  }

  // 2. Compute the mean stress for each node
  real meanStress = (localStresses[0] + localStresses[1] + localStresses[2]) / 3.0f;

  // 3. Compute deviatoric stress tensor
  #pragma unroll
  for (int i = 0; i < 3; ++i) {
    localStresses[i] -= meanStress;
  }

  // 4. Compute the second invariant for each node
  real tau = 0.5 * (localStresses[0] * localStresses[0] +
                    localStresses[1] * localStresses[1] +
                    localStresses[2] * localStresses[2]);
  tau += (localStresses[3] * localStresses[3] +
          localStresses[4] * localStresses[4] +
          localStresses[5] * localStresses[5]);
  tau = squareRoot(tau);

  // 5. Compute the plasticity criteria
  const real cohesionTimesCosAngularFriction = plasticity[blockIdx.x].cohesionTimesCosAngularFriction;
  const real sinAngularFriction = plasticity[blockIdx.x].sinAngularFriction;
  real taulim = cohesionTimesCosAngularFriction - meanStress * sinAngularFriction;
  taulim = maxValue(static_cast<real>(0.0), taulim);

  __shared__ unsigned isAdjusted;
  if (threadIdx.x == 0) { isAdjusted = static_cast<unsigned>(false); }
  __syncthreads();

  // 6. Compute the yield factor
  real factor = 0.0;
  if (tau > taulim) {
    isAdjusted = static_cast<unsigned >(true);
    factor = ((taulim / tau) - 1.0) * oneMinusIntegratingFactor;
  }

  // 7. Adjust deviatoric stress tensor if a node within a node exceeds the elasticity region
  __syncthreads();
  if (isAdjusted) {
    #pragma unroll
    for (int i = 0; i < NUM_STRESS_COMPONENTS; ++i) {
      elementTensors[threadIdx.x + elementTensorsColumn * i] = localStresses[i] * factor;
    }
  }

  if (threadIdx.x == 0) {
    isAdjustableVector[blockIdx.x] = isAdjusted;
  }
}

void adjustDeviatoricTensors(real **nodalStressTensors,
                             unsigned *isAdjustableVector,
                             const PlasticityData *plasticity,
                             const double oneMinusIntegratingFactor,
                             const size_t numElements,
                             void *streamPtr) {
  constexpr unsigned numNodes = tensor::QStressNodal::Shape[0];
  dim3 block(numNodes, 1, 1);
  dim3 grid(numElements, 1, 1);
  auto stream = reinterpret_cast<hipStream_t>(streamPtr);
  kernel_adjustDeviatoricTensors<<<grid, block, 0, stream>>>(nodalStressTensors,
                                                             isAdjustableVector,
                                                             plasticity,
                                                             oneMinusIntegratingFactor);
}


//--------------------------------------------------------------------------------------------------
__global__ void kernel_computePstrains(real **pstrains,
                                       const unsigned *isAdjustableVector,
                                       const real **modalStressTensors,
                                       const real *firsModes,
                                       const PlasticityData *plasticity,
                                       const double oneMinusIntegratingFactor,
                                       const double timeStepWidth,
                                       const double T_v,
                                       const size_t numElements) {
  // compute element id
  size_t index = threadIdx.y + blockIdx.x * blockDim.y;
  if ((isAdjustableVector[index]) && (index < numElements)) {
    // NOTE: Six threads (x-dimension) work on the same element.

    // get local data
    real *localPstrains = pstrains[index];
    const real *localModalTensor = modalStressTensors[index];
    const real *localFirstMode = &firsModes[NUM_STRESS_COMPONENTS * index];
    const PlasticityData *localData = &plasticity[index];

    constexpr auto elementTensorsColumn = leadDim<init::QStressNodal>();
    real factor = localData->mufactor / (T_v * oneMinusIntegratingFactor);
    real duDtPstrain = factor * (localFirstMode[threadIdx.x] - localModalTensor[threadIdx.x * elementTensorsColumn]);
    localPstrains[threadIdx.x] += timeStepWidth * duDtPstrain;

    __shared__ real squaredDuDtPstrains[NUM_STRESS_COMPONENTS];
    real coefficient = threadIdx.x < 3 ? static_cast<real>(0.5) : static_cast<real>(1.0);
    squaredDuDtPstrains[threadIdx.x] = coefficient * duDtPstrain * duDtPstrain;
    __syncthreads();

    if (threadIdx.x == 0) {
      real sum = 0.0;

      #pragma unroll
      for (int i = 0; i < NUM_STRESS_COMPONENTS; ++i) {
        sum += squaredDuDtPstrains[i];
      }
      localPstrains[6] += (timeStepWidth * squareRoot(duDtPstrain));
    }
  }
}


void computePstrains(real **pstrains,
                     const unsigned *isAdjustableVector,
                     const real **modalStressTensors,
                     const real *firsModes,
                     const PlasticityData *plasticity,
                     const double oneMinusIntegratingFactor,
                     const double timeStepWidth,
                     const double T_v,
                     const size_t numElements,
                     void *streamPtr) {
  dim3 block(NUM_STRESS_COMPONENTS, 32, 1);
  size_t numBlocks = (numElements + block.y - 1) / block.y;
  dim3 grid(numBlocks, 1, 1);
  auto stream = reinterpret_cast<hipStream_t>(streamPtr);
  kernel_computePstrains<<<grid, block, 0, stream>>>(pstrains,
                                                     isAdjustableVector,
                                                     modalStressTensors,
                                                     firsModes,
                                                     plasticity,
                                                     oneMinusIntegratingFactor,
                                                     timeStepWidth,
                                                     T_v,
                                                     numElements);
}
} // namespace plasticity
} // namespace aux
} // namespace device
} // namespace kernels
} // namespace seissol
